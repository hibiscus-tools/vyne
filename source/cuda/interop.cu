#include "cuda/interop.cuh"

hipTextureObject_t cuda_import_vulkan_texture(const vk::Device &device, const littlevk::Image &image)
{
	int fd = littlevk::find_memory_fd(device, image.memory);

	hipExternalMemoryHandleDesc memory_handle {};
	memory_handle.type = hipExternalMemoryHandleTypeOpaqueFd;
	memory_handle.handle.fd = fd;
	memory_handle.size = image.device_size();

	hipExternalMemory_t external_memory {};
	hipImportExternalMemory(&external_memory, &memory_handle);

	cudaExternalMemoryMipmappedArrayDesc mipmap_description {};
	mipmap_description.offset = 0;
	mipmap_description.numLevels = 1;
	mipmap_description.formatDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	mipmap_description.extent = make_hipExtent(image.extent.width, image.extent.height, 1);

	hipMipmappedArray_t mipmap_array = nullptr;
	cudaExternalMemoryGetMappedMipmappedArray(&mipmap_array, external_memory, &mipmap_description);

	hipResourceDesc resource_description {};
	resource_description.resType = hipResourceTypeMipmappedArray;
	resource_description.res.mipmap.mipmap = mipmap_array;

	hipTextureDesc texture_description {};
	texture_description.normalizedCoords = true;
	texture_description.readMode = hipReadModeNormalizedFloat;
	texture_description.filterMode = hipFilterModeLinear;

	hipTextureObject_t texture {};
	hipCreateTextureObject(&texture, &resource_description, &texture_description, nullptr);

	return texture;
}

hipSurfaceObject_t cuda_import_vulkan_surface(const vk::Device &device, const littlevk::Image &image)
{
	hipError_t error;

	int fd = littlevk::find_memory_fd(device, image.memory);

	hipExternalMemoryHandleDesc memory_handle {};
	memory_handle.type = hipExternalMemoryHandleTypeOpaqueFd;
	memory_handle.handle.fd = fd;
	memory_handle.size = image.device_size();

	hipExternalMemory_t external_memory {};
	error = hipImportExternalMemory(&external_memory, &memory_handle);
	cuda_check(error);

	cudaExternalMemoryMipmappedArrayDesc mipmap_description {};
	mipmap_description.offset = 0;
	mipmap_description.numLevels = 1;
	mipmap_description.formatDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	mipmap_description.extent = make_hipExtent(image.extent.width, image.extent.height, 0);

	hipMipmappedArray_t mipmap_array = nullptr;
	error = cudaExternalMemoryGetMappedMipmappedArray(&mipmap_array, external_memory, &mipmap_description);
	cuda_check(error);

	hipArray_t level_array = nullptr;
	error = hipGetMipmappedArrayLevel(&level_array, mipmap_array, 0);
	cuda_check(error);

	hipResourceDesc resource_description {};
	resource_description.resType = hipResourceTypeArray;
	resource_description.res.array.array = level_array;

	hipSurfaceObject_t surface {};
	error = hipCreateSurfaceObject(&surface, &resource_description);
	cuda_check(error);

	return surface;
}
